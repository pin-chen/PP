
#include <hip/hip_runtime.h>
__global__ void convolution(int filterWidth, float *filter, int imageHeight, int imageWidth, float *inputImage, float *outputImage)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int halffilterSize = filterWidth / 2;
    float sum = 0.0f;
    int k, l;
    int outer_begin = i - halffilterSize >= 0 ? -halffilterSize : -i;
    int outer_end = i + halffilterSize < imageHeight ? halffilterSize : imageHeight - i - 1;
    int inner_begin = j - halffilterSize >= 0 ? -halffilterSize : -j;
    int inner_end = j + halffilterSize < imageWidth ? halffilterSize : imageWidth - j - 1;
    for (k = outer_begin; k <= outer_end; k++)
    {
        for (l = inner_begin; l <= inner_end; l++)
        {
            sum += inputImage[(i + k) * imageWidth + j + l] *
                    filter[(k + halffilterSize) * filterWidth +
                            l + halffilterSize];
        }
    }
    outputImage[i * imageWidth + j] = sum;
}


extern "C" void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
                       float *inputImage, float *outputImage)
{
    float *filter_mem, *inputImage_mem, *outputImage_mem;
    int filterSize = filterWidth * filterWidth * sizeof(float);
    int dataSize = imageHeight * imageWidth * sizeof(float);

    hipMalloc(&filter_mem, filterSize);
    hipMalloc(&inputImage_mem, dataSize);
    hipMalloc(&outputImage_mem, dataSize);

    hipMemcpy(filter_mem, filter, filterSize, hipMemcpyHostToDevice);
    hipMemcpy(inputImage_mem, inputImage, dataSize, hipMemcpyHostToDevice);

    int block_size = 16;

    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid(imageWidth / block_size, imageHeight / block_size);

    convolution<<<dimBlock, dimGrid>>>(filterWidth, filter_mem, imageHeight, imageWidth, inputImage_mem, outputImage_mem);
    
    hipMemcpy(outputImage, outputImage_mem, dataSize, hipMemcpyDeviceToHost);
    
    hipFree(outputImage_mem);
    hipFree(inputImage_mem);
    hipFree(filter_mem);
}