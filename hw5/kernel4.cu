#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void mandelKernel(int* img, float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int count) {
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    if (thisX >= resX || thisY >= resY) return;
    
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;

    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    img[thisX + thisY * resX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);
    int* img_d;
    hipMalloc((void**)&img_d, size);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((int)((float) resX / dimBlock.x + 0.5), (int)((float) resY / dimBlock.y + 0.5));

    mandelKernel<<<dimGrid, dimBlock>>>(img_d, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations);

    hipDeviceSynchronize();

    hipMemcpy(img, img_d, size, hipMemcpyDeviceToHost);

    hipFree(img_d);
}
