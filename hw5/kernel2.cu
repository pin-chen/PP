#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void mandelKernel(int* img, float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int count, size_t pitch) {
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    if (thisX >= resX || thisY >= resY) return;
    // To avoid error caused by the floating number, use the following pseudo code
    //
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;

    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    ((int*)((int8_t*) img + thisY * pitch))[thisX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size_h = resX * resY * sizeof(int);
    int size_d = resX * sizeof(int);
    int* img_h;
    int* img_d;
    size_t pitch;

    hipHostAlloc(&img_h, size_h, hipHostMallocDefault);
    hipMallocPitch(&img_d, &pitch, size_d, resY);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((int)((float) resX / dimBlock.x + 0.5), (int)((float) resY / dimBlock.y + 0.5));

    mandelKernel<<<dimGrid, dimBlock>>>(img_d, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations, pitch);

    hipDeviceSynchronize();

    hipMemcpy2D(img_h, size_d, img_d, pitch, size_d, resY, hipMemcpyDeviceToHost);
    hipMemcpy(img, img_h, size_h, hipMemcpyHostToHost);

    hipHostFree(img_h);
    hipFree(img_d);
}
