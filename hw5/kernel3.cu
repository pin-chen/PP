#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define GROUP_SIZE 2

__global__ void mandelKernel(int* img, float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int count, size_t pitch) {
    int thisX = GROUP_SIZE * (blockIdx.x * blockDim.x + threadIdx.x);
    int thisY = GROUP_SIZE * (blockIdx.y * blockDim.y + threadIdx.y);

    int i, j;
    for (i = 0; i < GROUP_SIZE; i++) 
    {
        for (j = 0; j < GROUP_SIZE; j++) 
        {
            int tmpX = thisX + i;
            int tmpY = thisY + j;
            if (tmpX >= resX || tmpY >= resY) 
                continue;

            float c_re = lowerX + tmpX * stepX;
            float c_im = lowerY + tmpY * stepY;

            float z_re = c_re, z_im = c_im;
            int k;
            for (k = 0; k < count; k++) 
            {
                if (z_re * z_re + z_im * z_im > 4.f) 
                    break;
                    
                float new_re = z_re * z_re - z_im * z_im;
                float new_im = 2.f * z_re * z_im;
                z_re = c_re + new_re;
                z_im = c_im + new_im;
            }

            ((int*)((int8_t*)img + (tmpY) * pitch))[tmpX] = k;
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size_h = resX * resY * sizeof(int);
    int size_d = resX * sizeof(int);
    int* img_h;
    int* img_d;
    size_t pitch;

    hipHostAlloc(&img_h, size_h, hipHostMallocDefault);
    hipMallocPitch(&img_d, &pitch, size_d, resY);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    auto PER_SIZE = BLOCK_SIZE * GROUP_SIZE;
    dim3 dimGrid((int)((float) resX / PER_SIZE + 0.5), (int)((float)resY / PER_SIZE + 0.5));

    mandelKernel<<<dimGrid, dimBlock>>>(img_d, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations, pitch);

    hipDeviceSynchronize();

    hipMemcpy2D(img_h, size_d, img_d, pitch, size_d, resY, hipMemcpyDeviceToHost);
    hipMemcpy(img, img_h, size_h, hipMemcpyHostToHost);

    hipHostFree(img_h);
    hipFree(img_d);
}
